#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,									// 高斯点数量
	const float2* points_xy,				// [P 2] 高斯点的 2D 坐标
	const float* depths,					// [P] 高斯点的 2D 投影深度
	const uint32_t* offsets,				// [P] 前缀和数组，指示每个高斯点实例化自己影响到的 tile 时的写入起始偏移
	uint64_t* gaussian_keys_unsorted,				// 输出: [num_rendered]  tile ID(前 32 位) + depth(后 32 位)
	uint32_t* gaussian_values_unsorted,				// 输出: [num_rendered]  影响这个 tile 的高斯点的编号
	int* radii,								// [P] 每个高斯点投影半径
	dim3 grid								// (gridX,gridY,1)——屏幕被划分成多少个 tile
) {
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0) {
		// Find this Gaussian's offset in buffer for writing keys/values.
		// off 表示当前这个高斯点实例化自己影响到的 tile 时的写入起始偏移
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		
		uint2 rect_min, rect_max;
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++) {
			for (int x = rect_min.x; x < rect_max.x; x++) {
				// grid.x 是一行的数量, y 是在第几行(从 0 开始), x 是在第几列(从 0 开始)
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;		// tile ID + depth
				gaussian_values_unsorted[off] = idx;	// off 这个 tile 是 idx 这个高斯点影响的
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,	// geomBuffer 扩容函数 & 返回指针 (要往里填东西)
	std::function<char* (size_t)> binningBuffer,	// binningBuffer 扩容函数 & 返回指针 (要往里填东西)
	std::function<char* (size_t)> imageBuffer,		// imageBuffer 扩容函数 & 返回指针 (要往里填东西)
	const int P,			// 高斯点数量
	const int D,			// sh 的阶数
	const int M,			// sh 系数的数量
	const float* background,				// 背景颜色
	const int width,						// 图像宽度
	const int height,						// 图像高度
	const float* means3D,					// 高斯点的 3D 坐标 [P 3]
	const float* shs,						// sh 系数 [P M D(3)]
	const float* colors_precomp,			// 预先计算好的 RGB 颜色 (若有)
	const float* opacities,					// 所有高斯体的不透明度
	const float* scales,					// 每个高斯体的尺度 (在 xyz 轴的缩放长度)
	const float scale_modifier,				// 控制高斯体们的尺寸, 缩放因子
	const float* rotations,					// 每个高斯体的旋转变量
	const float* cov3D_precomp,				// 预先计算好的协方差矩阵 (若有)
	const float* viewmatrix,				// 视图矩阵
	const float* projmatrix,				// 投影矩阵
	const float* cam_pos,					// 相机在世界里的坐标
	const float tan_fovx,					// 单位深度处的半宽度
	const float tan_fovy,					// 单位深度处的半高度
	const bool prefiltered,					// 表示你是否已经在别的地方对颜色做过“预滤波”（模糊、降采样）处理。这里设为 False，让 rasterizer 自己来处理
	float* out_color,			// 渲染图像 (要往里填东西)
	float* depth,				// 反深度图 (要往里填东西)
	bool antialiasing,						// 是否开启抗锯齿
	int* radii,					// 每个高斯点投影半径 (要往里填东西)
	bool debug,								// 是否开启 debug 模式
	float* out_accum_alpha,// 每个 pixel 的剩余透射率 (要往里填东西)
	float* gauss_sum,
	int* gauss_count,
	int* last_contr_gauss,
	float* out_depths
) {
	// 想象一个针孔相机, 光线从物体经过针孔, 打到后面的成像平面, 焦距就是针孔到成像平面之间的距离
	// f 越大, 成像平面离针孔远，投影的物体看上去“更大”、视野更窄（长焦）。
	// f 越小, 成像平面离针孔近，投影的物体看上去“更小”、视野更宽（广角）。
	// 下面计算的 focal_x/focal_y 就是在针孔相机的放射角固定的前提下, 为了使得成像平面大小是 height x width 的 x/y 焦距
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	// required<GeometryState>(P) 是一个编译时/运行时帮助函数, 用来计算如果要存下 P 个高斯点的 GeometryState 需要的字节大小
	// GeometryState 是一个结构体, 里面存放了高斯点的各种信息: 坐标协方差颜色不透明度等
	size_t chunk_size = required<GeometryState>(P);

	// 在 geometryBuffer 中申请 chunk_size 大小的内存, 并返回指向这块内存的指针, 用来存放 P 个点的 GeometryState
	char* chunkptr = geometryBuffer(chunk_size);

	// 简单理解就行, fromChunk 是一个静态方法, 它接受一个原始内存指针 chunkptr 和高斯点数量 P, 然后返回一个把原始内存结构化好的 GeometryState 对象
	// 后续往里填东西就行
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	// 正常 radii 传进来的是一块区域内存的指针, 占位用的, 待填充
	// 可是如果 radii 没分配内存, 那么就用 geomState.internal_radii 这个指针
	if (radii == nullptr) {
		radii = geomState.internal_radii;
	}

	// 这里名字起的比较诡异, 这里的 block 其实就是说一个 block 分配 16x16x1 个 thread
	dim3 block(BLOCK_X, BLOCK_Y, 1);
	// 这里计算所需的 blocks 数量 (注意这里第一维是列)
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	// required<ImageState>(width * height) 是一个编译时/运行时帮助函数, 用来计算如果要存下 width * height 个像素的 ImageState 需要的字节大小
	size_t img_chunk_size = required<ImageState>(width * height);
	// 在 imageBuffer 中申请 img_chunk_size 大小的内存, 并返回指向这块内存的指针, 用来存放 width * height 个像素的 ImageState
	char* img_chunkptr = imageBuffer(img_chunk_size);
	// 简单理解就行, fromChunk 是一个静态方法, 它接受一个原始内存指针 img_chunkptr 和像素数量 width * height, 然后返回一个把原始内存结构化好的 ImageState 对象
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	/*
		到了这里, 我们已经有:
			GeometryState geomState: 里面存放了高斯点的各种信息: 坐标协方差颜色不透明度等
			ImageState imgState: 里面存放了像素的各种信息: 像素颜色、深度等
	*/

	// 如果不是在做标准的 RGB 渲染, 那么就需要提供预先计算好的颜色. 否则报错
	if (NUM_CHANNELS != 3 && colors_precomp == nullptr) {
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	// 把高斯体从世界坐标下的参数转换到屏幕空间下的各种中间量, 为后续真正的光栅化 render 做准备
	CHECK_CUDA(
		FORWARD::preprocess(
			P,							// 高斯点数量
			D,							// sh 的阶数
			M,							// sh 系数的数量
			means3D,					// 高斯点的 3D 坐标 [P 3]
			(glm::vec3*)scales,			// 每个高斯体的尺度 (在 xyz 轴的缩放长度) (这里的 glm::vec3* 意思是把原本 float* 指向一大片连续浮点数的指针, 解释为 glm::vec3*, 即在代码里可以直接写 scales[i].x)
			scale_modifier,				// 控制高斯体们的尺寸, 缩放因子
			(glm::vec4*)rotations,		// 每个高斯体的旋转变量
			opacities,					// 所有高斯体的不透明度
			shs,						// sh 系数 [P M D(3)]
			geomState.clamped,						// [P 3] 每个高斯点的 R/G/B 通道的值是否被 clamped 的标志位 (待写入)
			cov3D_precomp,				// 预先计算好的协方差矩阵 (若有)
			colors_precomp,				// 预先计算好的 RGB 颜色 (若有)
			viewmatrix,					// 视图矩阵
			projmatrix,					// 投影矩阵
			(glm::vec3*)cam_pos,		// 相机在世界里的坐标
			width,						// 图像宽度
			height,						// 图像高度
			focal_x,					// x 轴焦距
			focal_y,					// y 轴焦距
			tan_fovx,					// 单位深度处的半宽度
			tan_fovy,					// 单位深度处的半高度
			radii,									// [P] 每个高斯点投影半径 (要往里填东西)
			geomState.means2D,						// [P 2] 输出的 2D 投影中心
			geomState.depths,						// [P] 输出的高斯投影深度
			geomState.cov3D,						// [P 6] 输出的协方差矩阵 (如果没预传 cov3D_precomp, 就输出到这里)
			geomState.rgb,							// [P 3] 输出的投影点 RGB 颜色 (如果没预传 colors_precomp, 就输出到这里)
			geomState.conic_opacity,				// 输出的 2D 协方差矩阵的逆矩阵 和 输入的透明度
			tile_grid,					// (gridX,gridY,1)——屏幕被划分成多少个 tile
			geomState.tiles_touched,				// [P] 输出的每个投影点影响到的 tile 数量
			prefiltered,				// 是否开启预滤波
			antialiasing,				// 是否开启抗锯齿
			out_depths
		),
		debug
	)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	// 下面就可以理解算出 geomState.tiles_touched 这个数组的前缀和数组, 存到 geomState.point_offsets 里
	// 作用就是后边用于给每个高斯点分配缓存区用的
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(
			geomState.scanning_space,
			geomState.scan_size,
			geomState.tiles_touched,
			geomState.point_offsets,
			P
		),
		debug
	)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	// 下面这段就是我们待会要进行多少次渲染, 渲染的总次数就是 num_rendered, 也就是 geomState.point_offsets[P - 1], 即所有高斯点会影响到的 tiles 数量的总和
	int num_rendered;
	CHECK_CUDA(hipMemcpy(
			&num_rendered,
			geomState.point_offsets + P - 1,
			sizeof(int),
			hipMemcpyDeviceToHost
		),
		debug
	);

	// required<BinningState>(num_rendered) 是一个编译时/运行时帮助函数, 用来计算如果要存下 num_rendered 个渲染操作的 BinningState 需要的字节大小
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	// 在 binningBuffer 中申请 binning_chunk_size 大小的内存, 并返回指向这块内存的指针, 用来存放 num_rendered 个渲染操作的 BinningState
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	// 简单理解就行, fromChunk 是一个静态方法, 它接受一个原始内存指针 binning_chunkptr 和渲染操作数量 num_rendered, 然后返回一个把原始内存结构化好的 BinningState 对象
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	// 并且处理 P 个高斯点, 去找出每个高斯点影响到的 tile, 然后统计下信息
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,											// 高斯点数量
		geomState.means2D,							// [P 2] 高斯点的 2D 坐标
		geomState.depths,							// [P] 高斯点的 2D 投影深度
		geomState.point_offsets,					// [P] 前缀和数组，指示每个高斯点实例化自己影响到的 tile 时的写入起始偏移
		binningState.point_list_keys_unsorted,				// 输出: [num_rendered]  tile ID(前 32 位) + depth(后 32 位)
		binningState.point_list_unsorted,					// 输出: [num_rendered]  影响这个 tile 的高斯点的编号
		radii,										// [P] 每个高斯点投影半径
		tile_grid									// (gridX,gridY,1)——屏幕被划分成多少个 tile
	)
	CHECK_CUDA(, debug)

	// 下面这段代码就是对前面的 binningState.point_list_keys_unsorted 和 binningState.point_list_unsorted 进行基数排序
	// 让它们最终按照 "tile 从小到大, 同一 tile 内再按 depth 从小到大" 的顺序排列
	// 结果存到 binningState.point_list_keys 和 binningState.point_list 里
	int bit = getHigherMsb(tile_grid.x * tile_grid.y);
	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	// 把 imgState.ranges 这个数组清零, 这个数组的作用是存储每个 tile 的高斯点的范围
	// 这里作者写的比较令人迷惑, 因为 imgState 定义的时候是以像素为单位定义的, 但是为什么这里又用来存储 tile 的涉及范围了呢?
	// 其实是这样的, imgState 里有三块同样大小为 width * height 的内存区域, 分别是 accum_alpha, n_contrib 和 ranges
		// accu_alpha 和 n_contrib 是用来存储每个像素的 alpha 和贡献值的
		// ranges (uint2 数组) 用来存储 tile 的涉及范围, 也就是说只有 tile_grid 的内存区域会被用到, 剩下的不会用到
		// 所以说把 ranges 记录在 imgState 只是为了方便, 其实是可以单独开辟一块内存的
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);
	// Identify start and end of per-tile workloads in sorted list
	// imgState.ranges[t] 中的 t 表示 tildID
	// 前面的 binningState.point_list_keys 不是已经排好序, 存储了 (tildeID, depth) 的键值对了吗?
	// imgState.ranges[t].x 表示 t 这个 tile 在 point_list_keys 中的起始位置
	// imgState.ranges[t].y 表示 t 这个 tile 在 point_list_keys 中的结束位置+1
	// 比如 point_list_keys = [(0, 1), (0, 2), (0, 3), (1, 0)], 那么 imgState.ranges[0].x = 0, imgState.ranges[0].y = 3 (左闭右开)
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges
		);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
			tile_grid,					// (gridX,gridY,1)——屏幕被划分成多少个 tile
			block,						// (16,16,1)——每个 tile 里有多少个 thread
			imgState.ranges,			// [tileID 2] tile 的涉及范围, 值是 point_list_keys 中的索引
			binningState.point_list,	// 对应着 point_list_keys 中负责的高斯投影圆心的索引
			width,						// 图像宽度
			height,						// 图像高度
			geomState.means2D,			// [P 2] 高斯投影圆心的 2D 坐标
			feature_ptr,				// [P 3] 高斯投影圆心的 RGB 颜色
			geomState.conic_opacity,	// 高斯投影椭圆的 2D 协方差矩阵的逆矩阵 + 3D 高斯体的不透明度
			imgState.accum_alpha,				// 每个 pixel 的剩余透射率 (要往里填东西)
			imgState.n_contrib,					// 实际影响到该 pixel 的高斯体实例数量 (要往里填东西)
			background,					// 背景颜色
			out_color,							// 渲染图像 (要往里填东西)
			geomState.depths,			// [P] 高斯投影深度
			depth,								// 反深度图 (要往里填东西)
			gauss_sum,
			gauss_count,
			last_contr_gauss,
			out_accum_alpha
		),
		debug
	)

	// 渲染动作的总次数
	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	// 基本尺寸参数
	const int P,				// 高斯点数量
	const int D,				// sh 的阶数
	const int M,				// sh 系数的数量
	const int R,				// 前向所有 (tileID, depth) pairs 的个数

	// 前向需要的常量输入 (只读)
	const float* background,			// 背景颜色
	const int width,					// 图像宽度
	const int height,					// 图像高度
	const float* means3D,				// 高斯点的 3D 坐标 [P 3]
	const float* shs,					// sh 系数 [P M D]
	const float* colors_precomp,		// 预先计算好的 RGB 颜色 (若有)
	const float* opacities,				// 所有高斯体的不透明度
	const float* scales,				// 每个高斯体的尺度 (在 xyz 轴的缩放长度)
	const float scale_modifier,			// 控制高斯体们的尺寸, 缩放因子
	const float* rotations,				// 每个高斯体的旋转变量
	const float* cov3D_precomp,			// 预先计算好的协方差矩阵 (若有)
	const float* viewmatrix,			// 视图矩阵
	const float* projmatrix,			// 投影矩阵
	const float* campos,				// 相机在世界里的坐标
	const float tan_fovx,				// 单位深度处的半宽度
	const float tan_fovy,				// 单位深度处的半高度

	// 前向算出的中间量
	const int* radii,				// [P] 每个高斯点投影半径
	char* geom_buffer,				// geomBuffer
	char* binning_buffer,			// binningBuffer
	char* img_buffer,				// imageBuffer

	// 从 Python 端传回的 loss 对 forward 输出的梯度
	const float* dL_dpix,			// loss 对渲染 RGB 图像的梯度 [3 H W]
	const float* dL_invdepths,		// loss 对反深度图的梯度 [1 H W]

	// 需要往里写入值的梯度
	float* dL_dmean2D,			// loss 对 ndc 空间的高斯点 2D 坐标的梯度 [P 2]
	float* dL_dconic,			// loss 对高斯点 2D 协方差逆矩阵的梯度 [P 2 2]
	float* dL_dopacity,			// loss 对高斯点不透明度的梯度 [P 1]
	float* dL_dcolor,			// loss 对高斯点 RGB 颜色的梯度 [P 3]
	float* dL_dinvdepth,		// loss 对每个高斯体投影深度 view.z 的梯度 [P 1]
	float* dL_dmean3D,			// loss 对高斯点 3D 坐标的梯度 [P 3]
	float* dL_dcov3D,			// loss 对预先计算好的协方差的矩阵 (若有) 的梯度 [P 6]
	float* dL_dsh,				// loss 对 sh 系数的梯度 [P M D]
	float* dL_dscale,			// loss 对每个高斯体的尺度 (在 xyz 轴的缩放长度) 的梯度 [P 3]
	float* dL_drot,				// loss 对每个高斯体的旋转变量的梯度 [P 4]

	// 额外开关
	bool antialiasing,	// 是否开启抗锯齿
	bool debug			// 是否开启 debug 模式
) {
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);			// 管理高斯点信息的地方
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);		// 放 (tildeID, depth), 对应高斯idx 的地方
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);	// 放像素信息的地方 (以及每个 tile 负责的 pairs 的范围)

	if (radii == nullptr) {
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);	// y 轴焦距
	const float focal_x = width / (2.0f * tan_fovx);	// x 轴焦距

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);		// 一共需要的 block 数量 (tile 数量)
	const dim3 block(BLOCK_X, BLOCK_Y, 1);	// 一个 block 有 16x16 个 thread

	// 下面这个函数就是把 dL_dpix, dL_invdepths 反传到几个输出身上, 过程需要用到一些前向过程已经计算出来的量
	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
			tile_grid,					// (gridX,gridY,1)——屏幕被划分成多少个 tile
			block,						// (16,16,1)——每个 tile 里有多少个 thread
			imgState.ranges,			// [tileID 2] tile 负责的 pairs 范围, 值是 point_list_keys 中的索引 (左闭右开)
			binningState.point_list,	// 对应着 point_list_keys 中高斯体的索引 idx
			width,						// 图像宽度
			height,						// 图像高度
			background,					// 背景颜色
			geomState.means2D,			// [P 2] 高斯投影圆心的 2D 坐标
			geomState.conic_opacity,	// 高斯投影椭圆的 2D 协方差矩阵的逆矩阵 + 3D 高斯体的不透明度
			color_ptr,					// [P 3] 高斯投影圆心的 RGB 颜色
			geomState.depths,			// [P] 高斯投影深度
			imgState.accum_alpha,		// [P] 每个 pixel 的剩余透射率
			imgState.n_contrib,			// 实际影响到该 pixel 的高斯体实例数量, 换句话说, 该像素光线上穿过的高斯体数量
			dL_dpix,					// loss 对渲染 RGB 图像的梯度 [3 H W]
			dL_invdepths,				// loss 对反深度图的梯度 [1 H W]
			(float3*)dL_dmean2D,				// 输出: loss 对 ndc 空间的高斯点 2D 坐标的梯度 [P 2]
			(float4*)dL_dconic,					// 输出: loss 对高斯点 2D 协方差逆矩阵的梯度 [P 2 2]
			dL_dopacity,						// 输出: loss 对高斯点不透明度的梯度 [P 1]
			dL_dcolor,							// 输出: loss 对高斯点 RGB 颜色的梯度 [P 3]
			dL_dinvdepth						// 输出: loss 对每个高斯体投影深度 view.z 的梯度 [P 1]
		),
		debug
	);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(
			P,							// 高斯点数量
			D,							// sh 的阶数
			M,							// sh 系数的数量
			(float3*)means3D,			// [P 3] 高斯点的 3D 坐标 [P 3]
			radii,						// [P] 每个高斯点投影半径
			shs,						// [P M D] sh 系数
			geomState.clamped,			// [P 3] 每个高斯点的 R/G/B 通道的值是否被 clamped 的标志位
			opacities,					// 所有高斯体的不透明度
			(glm::vec3*)scales,			// [P 3] 每个高斯体的尺度 (在 xyz 轴的缩放长度)
			(glm::vec4*)rotations,		// [P 4] 每个高斯体的旋转变量
			scale_modifier,				// 控制高斯体们的尺寸, 缩放因子
			cov3D_ptr,					// 3D 协方差矩阵
			viewmatrix,					// 视图矩阵
			projmatrix,					// 投影矩阵	
			focal_x,					// x 轴焦距
			focal_y,					// y 轴焦距
			tan_fovx,					// 单位深度处的半宽度
			tan_fovy,					// 单位深度处的半高度
			(glm::vec3*)campos,			// 相机在世界里的坐标
			(float3*)dL_dmean2D,		// loss 对 ndc 空间的高斯点 2D 坐标的梯度 [P 2]
			dL_dconic,					// loss 对高斯点 2D 协方差逆矩阵的梯度 [P 2 2]
			dL_dinvdepth,				// loss 对每个高斯体投影深度 view.z 的梯度 [P 1]
			dL_dopacity,				// loss 对高斯点不透明度的梯度 [P 1]
			(glm::vec3*)dL_dmean3D,				// 输出: loss 对高斯点 3D 坐标的梯度 [P 3]
			dL_dcolor,					// loss 对高斯点 RGB 颜色的梯度 [P 3]
			dL_dcov3D,							// 输出: loss 对高斯体 3D 协方差的梯度 [P 6]
			dL_dsh,								// 输出: loss 对 sh 系数的梯度 [P M D]
			(glm::vec3*)dL_dscale,				// 输出: loss 对每个高斯体的尺度 (在 xyz 轴的缩放长度) 的梯度 [P 3]
			(glm::vec4*)dL_drot,				// 输出: loss 对每个高斯体的旋转变量的梯度 [P 4]
			antialiasing	// 是否开启抗锯齿
		),
		debug
	);
}
